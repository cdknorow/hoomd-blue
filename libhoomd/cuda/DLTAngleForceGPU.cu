#include "hip/hip_runtime.h"
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2009-2014 The Regents of
the University of Michigan All rights reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: dnlebard

#include "DLTAngleForceGPU.cuh"
#include "TextureTools.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

// SMALL a relatively small number
#define SMALL Scalar(0.001)

/*! \file DLTAngleForceGPU.cu
    \brief Defines GPU kernel code for calculating the DLT angle forces. Used by DLTAngleForceComputeGPU.
*/

//! Texture for reading angle parameters
scalar4_tex_t angle_params_tex4;
scalar2_tex_t angle_params_tex2;

//! Kernel for caculating DLT angle forces on the GPU
/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch Pitch of 2D virial array
    \param N number of particles
    \param d_pos device array of particle positions
    \param d_params Parameters for the angle force
    \param box Box dimensions for periodic boundary condition handling
    \param alist Angle data to use in calculating the forces
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
*/
extern "C" __global__ void gpu_compute_dlt_angle_forces_kernel(Scalar4* d_force,
                                                                    Scalar* d_virial,
                                                                    const unsigned int virial_pitch,
                                                                    const unsigned int N,
                                                                    const Scalar4 *d_pos,
                                                                    const Scalar2 *d_params_k,
                                                                    const Scalar4 *d_params_b,
                                                                    BoxDim box,
                                                                    const group_storage<3> *alist,
                                                                    const unsigned int *apos_list,
                                                                    const unsigned int pitch,
                                                                    const unsigned int *n_angles_list)
    {
    // start by identifying which particle we are to handle
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    int n_angles = n_angles_list[idx];

    // read in the position of our b-particle from the a-b-c triplet. (MEM TRANSFER: 16 bytes)
    Scalar4 idx_postype = d_pos[idx];  // we can be either a, b, or c in the a-b-c triplet
    Scalar3 idx_pos = make_scalar3(idx_postype.x, idx_postype.y, idx_postype.z);
    Scalar3 a_pos,b_pos; // allocate space for the a and atom in the a-b-c triplet

    // initialize the force to 0
    Scalar4 force_idx = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));

    Scalar3 force_vector;

    // initialize the virial to 0
    Scalar virial[6];
    for (int i = 0; i < 6; i++)
        virial[i] = Scalar(0.0);

    // loop over all angles
    for (int angle_idx = 0; angle_idx < n_angles; angle_idx++)
        {
        group_storage<3> cur_angle = alist[pitch*angle_idx + idx];

        int cur_angle_x_idx = cur_angle.idx[0];
        int cur_angle_type = cur_angle.idx[2];

        int cur_angle_abc = apos_list[pitch*angle_idx + idx];

        // get the a-particle's position (MEM TRANSFER: 16 bytes)
        Scalar4 x_postype = d_pos[cur_angle_x_idx];
        Scalar3 x_pos = make_scalar3(x_postype.x, x_postype.y, x_postype.z);
        // if curr_angle == 2, this is a dummy particle and we don't calculate anything
        if (cur_angle_abc != 2){
             // if curr_angle == 0 the b values are in order,
            if (cur_angle_abc == 0)
                {
                a_pos = idx_pos;
                b_pos = x_pos;
                }
            // if curr_angle == 1 the b values are backwards,
            if (cur_angle_abc == 1)
                {
                b_pos = idx_pos;
                a_pos = x_pos;
                }
 
            // calculate dr for a-b,c-b,and a-c
            Scalar3 dx = a_pos - b_pos;

            // apply periodic boundary conditions
            dx = box.minImage(dx);

            // get the angle parameters (MEM TRANSFER: 8 bytes)
            Scalar2 params_k = texFetchScalar2(d_params_k, angle_params_tex2, cur_angle_type);
            Scalar4 params_b = texFetchScalar4(d_params_b, angle_params_tex4, cur_angle_type);
            Scalar K1 = params_k.x;
            Scalar K2 = params_k.y;
            Scalar b_x = params_b.x;
            Scalar b_y = params_b.y;
            Scalar b_z = params_b.z;

            //Calculate ax ay az and a
            Scalar ax = dx.x + b_x;
            Scalar ay = dx.y + b_y;
            Scalar az = dx.z + b_z; 
            Scalar a  = ax * b_x + ay * b_y + az * b_z;

            // Force F1
            force_vector.x = - K1 * ax;
            force_vector.y = - K1 * ay;
            force_vector.z = - K1 * az;

            //Compute V1 Energy
            Scalar bond_eng = Scalar(0.5) * K1 * ( ax * ax + ay * ay + az * az );
       
            // Force F2
            force_vector.x += - K2 * b_x * a;
            force_vector.y += - K2 * b_y * a;
            force_vector.z += - K2 * b_z * a;

            // Compute V2 Energy
            bond_eng += Scalar(0.5) * K2 * (  ax * ax * b_x * b_x +
                                              ay * ay * b_y * b_y +
                                              az * az * b_z * b_z +
                                              2 * ax * b_x * ay * b_y +
                                              2 * ax * b_x * az * b_z +
                                              2 * az * b_z * ay * b_y);

            bond_eng *=  Scalar(0.5);
                
            // compute 1/2 of the virial, 1/2 for each atom in the bond
            // upper triangular version of virial tensor
            Scalar bond_virial[6];
            bond_virial[0] = Scalar(0.5) * dx.x * force_vector.x; // Fx*x
            bond_virial[1] = Scalar(0.5) * dx.y * force_vector.x; // Fx*y
            bond_virial[2] = Scalar(0.5) * dx.z * force_vector.x; // Fx*z
            bond_virial[3] = Scalar(0.5) * dx.y * force_vector.y; // Fy*y
            bond_virial[4] = Scalar(0.5) * dx.z * force_vector.y; // Fy*z

            if (cur_angle_abc == 0)
                {
                force_idx.x += force_vector.x;
                force_idx.y += force_vector.y;
                force_idx.z += force_vector.z;
                }
            if (cur_angle_abc == 1)
                {
                force_idx.x -= force_vector.x;
                force_idx.y -= force_vector.y;
                force_idx.z -= force_vector.z;
                }

            force_idx.w += bond_eng;

            for (int i = 0; i < 6; i++)
                virial[i] += bond_virial[i];
            }
        }
    // now that the force calculation is complete, write out the result (MEM TRANSFER: 20 bytes)
    d_force[idx] = force_idx;
    for (int i = 0; i < 6; i++)
        d_virial[i*virial_pitch+idx] = virial[i];
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial arary
    \param N number of particles
    \param d_pos device array of particle positions
    \param box Box dimensions (in GPU format) to use for periodic boundary conditions
    \param atable List of angles stored on the GPU
    \param pitch Pitch of 2D angles list
    \param n_angles_list List of numbers of angles stored on the GPU
    \param d_params K and t_0 params packed as Scalar2 variables
    \param n_angle_types Number of angle types in d_params
    \param block_size Block size to use when performing calculations
    \param compute_capability Device compute capability (200, 300, 350, ...)

    \returns Any error code resulting from the kernel launch
    \note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()

    \a d_params should include one Scalar2 element per angle type. The x component contains K the spring constant
    and the y component contains t_0 the equilibrium angle.
*/
hipError_t gpu_compute_dlt_angle_forces(Scalar4* d_force,
                                              Scalar* d_virial,
                                              const unsigned int virial_pitch,
                                              const unsigned int N,
                                              const Scalar4 *d_pos,
                                              const BoxDim& box,
                                              const group_storage<3> *atable,
                                              const unsigned int *apos_list,
                                              const unsigned int pitch,
                                              const unsigned int *n_angles_list,
                                              Scalar2 *d_params_k,
                                              Scalar4 *d_params_b,
                                              unsigned int n_angle_types,
                                              int block_size,
                                              const unsigned int compute_capability)
    {
    assert(d_params);

    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_compute_dlt_angle_forces_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    // setup the grid to run the kernel
    dim3 grid( N / run_block_size + 1, 1, 1);
    dim3 threads(run_block_size, 1, 1);
    // bind the texture on pre sm 35 arches
    if (compute_capability < 350)
        {
        hipError_t error = hipBindTexture(0, angle_params_tex4, d_params_b, sizeof(Scalar4) * n_angle_types);
        if (error != hipSuccess)
            return error;
        hipError_t error2 = hipBindTexture(0, angle_params_tex2, d_params_k, sizeof(Scalar2) * n_angle_types);
        if (error2 != hipSuccess)
            return error2;
        }

    // run the kernel
    gpu_compute_dlt_angle_forces_kernel<<< grid, threads>>>(d_force, d_virial, virial_pitch, N, d_pos, d_params_k, d_params_b, box,
        atable, apos_list, pitch, n_angles_list);

    return hipSuccess;
    }

